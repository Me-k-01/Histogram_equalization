#include "hip/hip_runtime.h"
#include "histoGPU.hpp"
#include "../utils/commonCUDA.hpp"
#include <iostream>


#define PI 3.14159265f

// fonction d'appel au fonction gpu
void gpuCall(Image & f_Image, int nbEchantillon){

    // Tailles 
    unsigned int sizeImage = f_Image._width * f_Image._height;
    unsigned int sizeTableInBytes = sizeImage * sizeof(float);
    unsigned int sizeImageInBytes = sizeImage * sizeof(unsigned char) * f_Image._nbChannels; 
    // Création des pointeurs pour gpu
    float *hueTable, *saturationTable, *valueTable;
    unsigned char * pixelTableIn, *pixelTableOut;
    unsigned int * histoTable, *repartTable;

    HANDLE_ERROR(hipMalloc((void**)&pixelTableIn, sizeImageInBytes));
    HANDLE_ERROR(hipMalloc((void**)&pixelTableOut, sizeImageInBytes));
    HANDLE_ERROR(hipMalloc((void**)&hueTable, sizeTableInBytes));
    HANDLE_ERROR(hipMalloc((void**)&saturationTable, sizeTableInBytes));
    HANDLE_ERROR(hipMalloc((void**)&valueTable, sizeTableInBytes));
    HANDLE_ERROR(hipMalloc((void**)&histoTable, nbEchantillon*sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void**)&repartTable, nbEchantillon*sizeof(unsigned int)));

    HANDLE_ERROR(hipMemcpy(pixelTableIn, f_Image._pixels, sizeImageInBytes, hipMemcpyHostToDevice));

    //définition des bloc et grille selon les différents Kernel
    dim3 blocRGB2HSV(32,32,1);
    dim3 grilleRGB2HSV((f_Image._width + blocRGB2HSV.x-1)/blocRGB2HSV.x,(f_Image._height + blocRGB2HSV.y-1)/blocRGB2HSV.y,1);
    
    dim3 blocHistogramme(32,1,1);
    dim3 grilleHistogramme(1,1,1);
    
    dim3 blocRepart(32,1,1);
    dim3 grilleRepart(1,1,1);
    
    dim3 blocEqualization(32,1,1);
    dim3 grilleEqualization(1,1,1);
    
    dim3 blocHSV2RGB(32,32,1);
    dim3 grilleHSV2RGB((f_Image._width + blocHSV2RGB.x-1)/blocHSV2RGB.x,(f_Image._height + blocHSV2RGB.y-1)/blocHSV2RGB.y,1);


    rgb2hsv<<<blocRGB2HSV, grilleRGB2HSV>>>(pixelTableIn, sizeImage, hueTable, saturationTable, valueTable);
    histogram<<<blocHistogramme, grilleHistogramme>>>(valueTable, sizeImage, nbEchantillon, histoTable);
    repart<<<blocRepart, grilleRepart>>>(histoTable, nbEchantillon, repartTable);
    equalization<<<blocEqualization,grilleEqualization>>>(repartTable, nbEchantillon, valueTable, sizeImage);

    hsv2rgb<<<blocHSV2RGB,grilleHSV2RGB>>>(hueTable,saturationTable,valueTable, sizeImage, pixelTableOut);

    HANDLE_ERROR(hipMemcpy(f_Image._pixels, pixelTableOut, sizeImageInBytes,hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(pixelTableIn));
    HANDLE_ERROR(hipFree(pixelTableOut));
    HANDLE_ERROR(hipFree(hueTable));
    HANDLE_ERROR(hipFree(saturationTable));
    HANDLE_ERROR(hipFree(valueTable));
    HANDLE_ERROR(hipFree(histoTable));
    HANDLE_ERROR(hipFree(repartTable));
}

// fonction d'appel au fonction gpu pour tests
void gpuCallTest(Image & f_Image, int nbEchantillon){

    // Tailles 
    unsigned int sizeImage = f_Image._width * f_Image._height;
    unsigned int sizeTableInBytes = sizeImage * sizeof(float);
    unsigned int sizeImageInBytes = sizeImage * sizeof(unsigned char) * f_Image._nbChannels; 
    // Création des pointeurs pour gpu
    float *hueTable, *saturationTable, *valueTable;
    unsigned char * pixelTableIn, *pixelTableOut;
    unsigned int * histoTable, *repartTable;
    

    HANDLE_ERROR(hipMalloc((void**)&pixelTableIn, sizeImageInBytes));
    HANDLE_ERROR(hipMalloc((void**)&pixelTableOut, sizeImageInBytes));
    HANDLE_ERROR(hipMalloc((void**)&hueTable, sizeTableInBytes));
    HANDLE_ERROR(hipMalloc((void**)&saturationTable, sizeTableInBytes));
    HANDLE_ERROR(hipMalloc((void**)&valueTable, sizeTableInBytes));
    HANDLE_ERROR(hipMalloc((void**)&histoTable, nbEchantillon*sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void**)&repartTable, nbEchantillon*sizeof(unsigned int)));

    HANDLE_ERROR(hipMemcpy(pixelTableIn, f_Image._pixels, sizeImageInBytes, hipMemcpyHostToDevice));


    //définition des bloc et grille selon les différents Kernel
    dim3 blocRGB2HSV(32,32,1);
    dim3 grilleRGB2HSV((f_Image._width + blocRGB2HSV.x-1)/blocRGB2HSV.x,(f_Image._height + blocRGB2HSV.y-1)/blocRGB2HSV.y,1);
    rgb2hsv<<<blocRGB2HSV, grilleRGB2HSV>>>(pixelTableIn, sizeImage, hueTable, saturationTable, valueTable);

        dim3 blocHistogramme(32,1,1);
        dim3 grilleHistogramme(1,1,1);
        histogram<<<blocHistogramme, grilleHistogramme>>>(valueTable, sizeImage, nbEchantillon, histoTable);


        dim3 blocRepart(32,1,1);
        dim3 grilleRepart(1,1,1);
        repart<<<blocRepart, grilleRepart>>>(histoTable, nbEchantillon, repartTable);

    for (int i = 1; i < 1025; i++)
    {
        
    dim3 blocEqualization(i,1,1);
    dim3 grilleEqualization(1,1,1);
        
    equalization<<<blocEqualization,grilleEqualization>>>(repartTable, nbEchantillon, valueTable, sizeImage);
    }
    
    
    //HANDLE_ERROR(hipMemcpy(f_Image._pixels, pixelTableOut, sizeImageInBytes,hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(pixelTableIn));
    HANDLE_ERROR(hipFree(pixelTableOut));
    HANDLE_ERROR(hipFree(hueTable));
    HANDLE_ERROR(hipFree(saturationTable));
    HANDLE_ERROR(hipFree(valueTable));
    HANDLE_ERROR(hipFree(histoTable));
    HANDLE_ERROR(hipFree(repartTable));
}

// Fonction qui pour chaque pixel de l’image, calcule sa valeur dans l’espace HSV, et répartit le résultat dans trois tableaux différents
__global__ void rgb2hsv(const unsigned char f_PixelTable[], unsigned int f_sizeTable, float f_HueTable[],float f_SaturationTable[],float f_ValueTable[]){
    int tidx = threadIdx.x + blockIdx.x*blockDim.x;
    int tidy = threadIdx.y + blockIdx.y*blockDim.y;
    int tidglobal = tidx + tidy *blockDim.x*gridDim.x;
    int nbThreadTotal = blockDim.x*gridDim.x*blockDim.y*gridDim.y;

    while (tidglobal < f_sizeTable)
    {
        float red = (float)f_PixelTable[tidglobal*3];
        float green = (float)f_PixelTable[tidglobal*3+1];
        float blue = (float)f_PixelTable[tidglobal*3+2];

        float colormax = fmaxf(red, fmaxf(green, blue));
        float colormin = fminf(red, fminf(green, blue));

        f_ValueTable[tidglobal] = colormax/255.0f;
        
        if(colormax > 0){
            f_SaturationTable[tidglobal] = 1.0f-(colormin/colormax);
        }
        else
        {
            f_SaturationTable[tidglobal] = 0.0f;
        }

        if(colormax - colormin > 0){

            float hue = (acosf((red - (green/2.0f + blue/2.0f))/sqrtf(red*red + green*green + blue*blue - (red*green + red*blue + green*blue))))*180/PI;
            if( blue > green){
                f_HueTable[tidglobal] = 360.0f - hue;
            }
            else{
                f_HueTable[tidglobal] = hue;
            }
        }
        else{
                f_HueTable[tidglobal] = 0.0f;
        }
        tidglobal += nbThreadTotal;
    }
    
}

// Transformation de HSV vers RGB (donc de trois tableaux vers un seul).
__global__ void hsv2rgb(const float f_HueTable[],const float f_SaturationTable[], const float f_ValueTable[], unsigned int f_sizeTable, unsigned char f_PixelTable[]){

    int tidx = threadIdx.x + blockIdx.x*blockDim.x;
    int tidy = threadIdx.y + blockIdx.y*blockDim.y;
    int tidglobal = tidx + tidy *blockDim.x*gridDim.x;
    int nbThreadTotal = blockDim.x*gridDim.x*blockDim.y*gridDim.y;


    while (tidglobal < f_sizeTable)
    {
        float colormax = 255.0f * f_ValueTable[tidglobal];
        float colormin = colormax*(1.0f-f_SaturationTable[tidglobal]);

        float h = f_HueTable[tidglobal];

        float z = (colormax - colormin)* (1.0f - fabsf(fmodf(h/60.0f,2.0f) -1.0f));

        if(h < 60){
            f_PixelTable[tidglobal*3] = (unsigned char)roundf(colormax);
            f_PixelTable[tidglobal*3+1] = ((unsigned char)roundf(z + colormin));
            f_PixelTable[tidglobal*3+2] = ((unsigned char)roundf(colormin));
        }
        else if (h < 120){
            f_PixelTable[tidglobal*3] = ((unsigned char)roundf(z + colormin));
            f_PixelTable[tidglobal*3+1] = ((unsigned char)roundf(colormax));
            f_PixelTable[tidglobal*3+2] = ((unsigned char)roundf(colormin));
        }
        else if (h < 180){
            f_PixelTable[tidglobal*3] = ((unsigned char)roundf(colormin));
            f_PixelTable[tidglobal*3+1] = ((unsigned char)roundf(colormax));
            f_PixelTable[tidglobal*3+2] = ((unsigned char)roundf(z + colormin));
        }
        else if (h < 240){
            f_PixelTable[tidglobal*3] = ((unsigned char)roundf(colormin));
            f_PixelTable[tidglobal*3+1] = ((unsigned char)roundf(z + colormin));
            f_PixelTable[tidglobal*3+2] = ((unsigned char)roundf(colormax));
        }
        else if (h < 300){
            f_PixelTable[tidglobal*3] = ((unsigned char)roundf(z + colormin));
            f_PixelTable[tidglobal*3+1] = ((unsigned char)roundf(colormin));
            f_PixelTable[tidglobal*3+2] = ((unsigned char)roundf(colormax));
        }
        else{
            f_PixelTable[tidglobal*3] = ((unsigned char)roundf(colormax));
            f_PixelTable[tidglobal*3+1] = ((unsigned char)roundf(colormin));
            f_PixelTable[tidglobal*3+2] = ((unsigned char)roundf(z + colormin));
        }
        tidglobal+=nbThreadTotal;
    }

}

// Fonction qui à partir de la composante V de chaque pixel, calcule l’histogramme de l’image.
__global__ void histogram(const float f_ValueTable[], unsigned int sizeTable, const unsigned int f_NbEchantillon, unsigned int f_HistoTable[]) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x ; // tidx
	for (; tidx < sizeTable; tidx += gridDim.x * blockDim.x) {
        int indexHist = roundf(f_ValueTable[tidx] * f_NbEchantillon);
        // On doit attendre que les threads ont terminer d'écrire sur la valeur pour incrémenter.
        atomicAdd(&f_HistoTable[indexHist], 1.f);
    }
}

// À partir de l’histogramme, applique la fonction de répartition r(l)
__global__ void repart(const unsigned int f_HistoTable[], const unsigned int sizeTable, unsigned int f_RepartionTable[]) {
    //__shared__ repartitionTable [sizeTable]; 

    int tidx = blockIdx.x * blockDim.x + threadIdx.x ; // tidx
	for (; tidx < sizeTable; tidx += gridDim.x * blockDim.x) { 
        // Deux façons de procéder:

        // On attend que la valeur précedente soit calculée avec de la synchronisation de thread
        //__syncthreads() ou atomicAdd
        //f_RepartionTable[x] = f_RepartionTable[x - 1] + f_HistoTable[x];

        // Soit on fait des calculs redondants de somme
        int res = 0;
        for (size_t k = 0; k <= tidx; k++) {  
            res += f_HistoTable[k]; 
        }
        f_RepartionTable[tidx] = res;
    } 
}

// À partir de la répartition précédente, “étaler” l’histogramme.
__global__ void equalization(const unsigned int f_RepartionTable[], unsigned int sizeTableRepartition, float f_ValueTable[], const unsigned int sizeValueTable) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x ; // tidx
    // sizeTableRepartition = L
    // sizeValueTable = n
    float coef = ((float)sizeTableRepartition - 1.f) / (float)(sizeValueTable * sizeTableRepartition) ; // (L - 1) / (L * n)
    sizeTableRepartition --; // avoir L-1 avant la boucle
    for (; tidx < sizeValueTable; tidx += gridDim.x * blockDim.x) {
        unsigned int indiceRepar = roundf(f_ValueTable[tidx] * sizeTableRepartition);
        f_ValueTable[tidx] = coef * f_RepartionTable[indiceRepar];
    }
}
